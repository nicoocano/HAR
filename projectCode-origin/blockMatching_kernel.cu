#pragma once
#include <hip/hip_runtime.h>

#ifndef BLOCK_MATCHING_KERNEL
#define BLOCK_MATCHING_KERNEL
#define INDXs(s,i,j)   ((s) * (i) + (j) + 0)

__device__ double computeMatchKernel(unsigned char *im,
		    int im_step,
		    unsigned char *bl,
		    int bl_step,
		    int bl_cols,
		    int bl_rows,
		    int oi, 
		    int oj, 
		    int stride){
  
  if (!im || !bl) return 0.0;

  double nb = (bl_cols*bl_rows);
  double x = 0;
  for(int i = 0;i < bl_rows-stride+1;i+= stride){
    for(int j = 0;j < bl_cols-stride+1;j+= stride){
      unsigned char v1 = im[INDXs(im_step,oi+i,oj+j)];
      unsigned char v2 = bl[INDXs(bl_step,i,j)];
      x += (v2-v1)*(v2-v1);
      //im[INDXs(im_step,oi+i,oj+j)] = ABS(v2-v1);
    }
  }
  x = x / nb;
  //  printf("%f\n",x);
  return x;
}

struct DataOut{
	double minVal;
	int coord_i_min;
	int coord_j_min;
};

__global__ void blockMatching_kernel(int jend,int stride,unsigned char* im, int im_step, unsigned char *bl, int bl_step,int bl_cols,int bl_rows, DataOut* result){
		
	__shared__ DataOut tab_data_out[1024];
	
	DataOut temp;
	temp.minVal=1000000000000000;// a changer	
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	for(int j = 0;j < jend-stride+1;j+=stride){
	  double x = computeMatchKernel(im,im_step,
				  bl,bl_step,bl_cols,bl_rows,
				  tid,j,stride);
	  
		 if(x<temp.minVal){
			 temp.minVal=x;
			 temp.coord_i_min=tid;
			 temp.coord_j_min=j;
		}
	}
			
	tab_data_out[tid]=temp;
	__syncthreads();
	
	for (unsigned int s = 1; s<blockDim.x * blockDim.y; s *=2){
		int index = 2 * s * threadIdx.x;
		
		if (index < blockDim.x * blockDim.y){

			if (tab_data_out[index].minVal > tab_data_out[index + s].minVal){
				if((index + s) < blockDim.x * blockDim.y){
					tab_data_out[index] = tab_data_out[index + s];			
				}	
			}
		}
		__syncthreads();
	}

	if(threadIdx.x==0){
		result[blockIdx.x] = tab_data_out[0];
	}


	
	  
}
#endif
